#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <vector>
#include <time.h> 

using namespace std;

struct Matrix
{
    int width;
    int height;
    float *elements;
};

const int width = 1200;
const int height = 1200;

float M1[height][width];
float M2[height][width];
float M3[height][width];

// 获取矩阵A的(row, col)元素
__device__ float getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

// 为矩阵A的(row, col)元素赋值
__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}

// 矩阵相乘kernel，2-D，每个线程计算一个元素
__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
	float Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < A->width; ++i)
	{
		Cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	setElement(C, row, col, Cvalue);
}

__host__ float hostgetElement(Matrix *A, int row, int col)
{
    cout << A->elements[1];
	return A->elements[row * A->width + col];
}

// 为矩阵A的(row, col)元素赋值
__host__ void hostsetElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}

int main()
{
    printf("Computing Complexity(Matrix dimension) is [%d, %d] X [%d, %d]\n", width, width, width, width);
    Matrix *A, *B, *C;
    // 申请托管内存
    clock_t start, end;
    start = clock();
    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));
    int nBytes = width * height * sizeof(float);
    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    // 初始化数据
    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;

    for (int i = 0; i < width * height; ++i)
    {
        float a = (4.0 * rand() / RAND_MAX) - 2.0;
        float b = (4.0 * rand() / RAND_MAX) - 2.0;
        A->elements[i] = a;
        B->elements[i] = b;
        M1[i%height][i%width] = a;
        M2[i%height][i%width] = b;
    }

    // 定义kernel的执行配置
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);
    // 执行kernel
    matMulKernel << < gridSize, blockSize >> >(A, B, C);

    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();

    // 并行计时
    end = clock();
    float t1 = (double)(end-start)/CLOCKS_PER_SEC;
    printf("Cuda program runs in %.2f seconds.\n", t1);

    float temp = 0;
    start = clock();
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
        {
            for (int k = 0; k < width; k++)
            {
                temp += M1[i][k] * M2[k][j];
            }
            M3[i][j] = temp;
            temp = 0;
        }
    }
    end = clock();
    float t2 = (double)(end-start)/CLOCKS_PER_SEC;
    printf("Serial program runs in %.2f seconds.\n", t2);
    printf("Speedup is %.2f\n", (t2/t1));
    return 0;
}